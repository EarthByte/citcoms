#include "hip/hip_runtime.h"
/* -*- C -*- */
/* vim:set ft=c: */

#if __CUDA_ARCH__ < 130
/* for double-precision floating-point */
#error This code requires compute capability 1.3 or higher; try giving "-arch sm_13".
#endif


#include "global_defs.h"
#include "element_definitions.h"
#include <assert.h>
#include <stdio.h>


enum {
    CAPS_PER_PROC = 1,
    M = 1, /* cap # */
    NSD = 3, /* Spatial extent: 3d */
    MAX_EQN = NSD*14,
};


struct Some_variables {
    
    struct /*MESH_DATA*/ {
        int NEQ;
        int NNO;
    } lmesh;
    
    struct ID *ID;
    
    higher_precision *Eqn_k[NSD+1];
    int *Node_map;
    
    double *BI;
    
    double *temp;
    unsigned int *NODE;
    
    int2 *term;
};


/*------------------------------------------------------------------------*/

static void construct_E(
    struct Some_variables **d_E,
    struct Some_variables *s_E, /* host's shadow copy of d_E */
    struct Some_variables *E
    )
{
    /* construct a copy of 'E' in device memory */
    
    int neq = E->lmesh.NEQ;
    int nno = E->lmesh.NNO;
    
    s_E->lmesh.NEQ = E->lmesh.NEQ;
    s_E->lmesh.NNO = E->lmesh.NNO;
    
    /* ID -- cf. allocate_common_vars()*/
    hipMalloc((void **)&s_E->ID, (nno+1)*sizeof(struct ID));
    hipMemcpy(s_E->ID, E->ID, (nno+1)*sizeof(struct ID), hipMemcpyHostToDevice);
    
    /* Eqn_k, Node_map -- cf. construct_node_maps() */
    size_t matrix = MAX_EQN * nno;
    s_E->Eqn_k[0] = 0;
    hipMalloc((void **)&s_E->Eqn_k[1], 3*matrix*sizeof(higher_precision));
    s_E->Eqn_k[2] = s_E->Eqn_k[1] + matrix;
    s_E->Eqn_k[3] = s_E->Eqn_k[2] + matrix;
    hipMemcpy(s_E->Eqn_k[1], E->Eqn_k[1], matrix*sizeof(higher_precision), hipMemcpyHostToDevice);
    hipMemcpy(s_E->Eqn_k[2], E->Eqn_k[2], matrix*sizeof(higher_precision), hipMemcpyHostToDevice);
    hipMemcpy(s_E->Eqn_k[3], E->Eqn_k[3], matrix*sizeof(higher_precision), hipMemcpyHostToDevice);
    hipMalloc((void **)&s_E->Node_map, matrix*sizeof(int));
    hipMemcpy(s_E->Node_map, E->Node_map, matrix*sizeof(int), hipMemcpyHostToDevice);
    
    /* BI -- cf. allocate_velocity_vars() */
    hipMalloc((void **)&s_E->BI, neq*sizeof(double));
    hipMemcpy(s_E->BI, E->BI, neq*sizeof(double), hipMemcpyHostToDevice);
    
    /* temp -- cf. allocate_velocity_vars() */
    hipMalloc((void **)&s_E->temp, (neq+1)*sizeof(double));
    hipMemcpy(s_E->temp, E->temp, (neq+1)*sizeof(double), hipMemcpyHostToDevice);
    
    /* NODE -- cf. allocate_common_vars() */
    hipMalloc((void **)&s_E->NODE, (nno+1)*sizeof(unsigned int));
    hipMemcpy(s_E->NODE, E->NODE, (nno+1)*sizeof(unsigned int), hipMemcpyHostToDevice);
    
    /* term */
    hipMalloc((void **)&s_E->term, (neq+1) * MAX_EQN * sizeof(int2));
    hipMemcpy(s_E->term, E->term, (neq+1) * MAX_EQN * sizeof(int2), hipMemcpyHostToDevice);
    
    /* E */
    hipMalloc((void**)d_E, sizeof(Some_variables));
    hipMemcpy(*d_E, s_E, sizeof(Some_variables), hipMemcpyHostToDevice);
    
    return;
}

static void destroy_E(
    struct Some_variables *d_E,
    struct Some_variables *s_E
    )
{
    hipFree(s_E->ID);
    hipFree(s_E->Eqn_k[1]);
    hipFree(s_E->Node_map);
    hipFree(s_E->BI);
    hipFree(s_E->temp);
    hipFree(s_E->NODE);
    hipFree(s_E->term);
    hipFree(d_E);
}


/*------------------------------------------------------------------------*/
/* from Element_calculations.c */

__global__ void n_assemble_del2_u(
    struct Some_variables *E,
    double *u, double *Au,
    int strip_bcs
    )
{
    int n = blockIdx.x + 1; /* 1 <= n <= E->lmesh.NNO */
    int doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    unsigned int tid = threadIdx.x; /* 0 <= tid < MAX_EQN */
    
    /* Each block writes one element of Au in global memory: Au[eqn]. */
    int eqn = E->ID[n].doff[doff]; /* XXX: Compute this value? */
    
    if (strip_bcs) {
        /* See get_bcs_id_for_residual(). */
        unsigned int flags = E->NODE[n];
        unsigned int vb = 0x1 << doff; /* VBX, VBY, or VBZ */
        if (flags & vb) {
            /* no-op: Au[eqn] is zero */
            if (tid == 0) {
                Au[eqn] = 0.0;
            }
            /* XXX: Hundreds of blocks exit here (E->num_zero_resid).
               Does it matter? */
            return;
        }
    }
    
    /* The partial sum computed by this thread. */
    double acc;
    
    /* Part I: The terms here are easily derived from the block and
       thread indices. */
    {
        int e = n; /* 1 <= e <= E->lmesh.NNO */
        int i = (int)tid; /* 0 <= i < MAX_EQN */
        
        if (i < 3) {
            acc = 0.0;
        } else {
            int *C = E->Node_map + (e-1)*MAX_EQN;
            higher_precision *B = E->Eqn_k[doff]+(e-1)*MAX_EQN;
            double UU = u[C[i]];
            acc = B[i]*UU;
        }
    }
    
    /* Part II: These terms are more complicated. */
    {
        int2 *term = E->term + eqn*MAX_EQN;
        int2 pair = term[tid];
        int e = pair.x; /* 1 <= e <= E->lmesh.NNO */
        int i = pair.y; /* 0 <= i < MAX_EQN */
        
        if (i != -1) {
            /* XXX: Compute these values? */
            int eqn1 = E->ID[e].doff[1];
            int eqn2 = E->ID[e].doff[2];
            int eqn3 = E->ID[e].doff[3];
            
            double U1 = u[eqn1];
            double U2 = u[eqn2];
            double U3 = u[eqn3];
            
            higher_precision *B1, *B2, *B3;
            B1 = E->Eqn_k[1]+(e-1)*MAX_EQN;
            B2 = E->Eqn_k[2]+(e-1)*MAX_EQN;
            B3 = E->Eqn_k[3]+(e-1)*MAX_EQN;
            
            acc += B1[i]*U1 +
                   B2[i]*U2 +
                   B3[i]*U3;
        } else {
            /* XXX: A considerable number of threads idle here. */
        }
    }
    
    /* Reduce the partial sums for this block.
       Based on reduce2() in the CUDA SDK. */
    __shared__ double sum[MAX_EQN];
    sum[tid] = acc;
    __syncthreads();
    for (unsigned int s = MAX_EQN/2; s > 0; s >>= 1) {
        if (tid < s) {
            sum[tid] += sum[tid + s];
        }
        /* XXX: not always necessary */
        __syncthreads();
    }
    
    /* Each block writes one element of Au in global memory. */
    if (tid == 0) {
        Au[eqn] = sum[0];
        
        if (n == 1 && doff == 1) {
            /* Well, actually, the first block writes one more. */
            Au[E->lmesh.NEQ] = 0.0;
        }
    }
    
    return;
}


/*------------------------------------------------------------------------*/
/* These are based on the function from General_matrix_functions.c. */

__global__ void gauss_seidel_0(
    struct Some_variables *E,
    double *d0,
    double *Ad
    )
{
    const double zeroo = 0.0;
    int i;
    
    i = blockIdx.x; /* 0 <= i < E->lmesh.NEQ */
    d0[i] = Ad[i] = zeroo;
}

__global__ void gauss_seidel_1(
    struct Some_variables *E,
    double *F, double *Ad
    )
{
    const double zeroo = 0.0;
    const int neq = E->lmesh.NEQ;
    
    int i, doff, eqn;
    
    i = blockIdx.x + 1; /* 1 <= i <= E->lmesh.NNO */
    doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    eqn = E->ID[i].doff[doff];
    
    if (E->NODE[i] & OFFSIDE) {
        E->temp[eqn] = (F[eqn] - Ad[eqn])*E->BI[eqn];
    } else {
        E->temp[eqn] = zeroo;
    }
    
    if (i == 1 && doff == 1) {
        E->temp[neq] = zeroo;
        Ad[neq] = zeroo;
    }
}

__global__ void gauss_seidel_2(
    struct Some_variables *E,
    double *F, double *Ad
    )
{
    int i, doff, eqn;
    
    i = blockIdx.x + 1; /* 1 <= i <= E->lmesh.NNO */
    doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    eqn = E->ID[i].doff[doff];
    
    int *C;
    higher_precision *B;
    double UU, Ad_eqn;
    int j;
    
    C = E->Node_map+(i-1)*MAX_EQN;
    B = E->Eqn_k[doff]+(i-1)*MAX_EQN;
    
    /* load from global memory */
    Ad_eqn = Ad[eqn];
    
    /* Ad on boundaries differs after the following operation, but
       no communications are needed yet, because boundary Ad will
       not be used for the G-S iterations for interior nodes */
    
    for (j=3;j<MAX_EQN;j++)  {
        UU = E->temp[C[j]];
        Ad_eqn += B[j]*UU;
    }
    
    /* store to global memory */
    Ad[eqn] = Ad_eqn;
    
    if (!(E->NODE[i] & OFFSIDE))   {
        E->temp[eqn] = (F[eqn] - Ad_eqn)*E->BI[eqn];
    }

}

__global__ void gauss_seidel_3(
    struct Some_variables *E,
    double *d0,
    double *Ad
    )
{
    int n = blockIdx.x + 1; /* 1 <= n <= E->lmesh.NNO */
    int doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    unsigned int tid = threadIdx.x; /* 0 <= tid < MAX_EQN */
    
    /* Each block writes one element of Ad and d0 in global memory:
       Ad[eqn], d0[eqn]. */
    int eqn = E->ID[n].doff[doff]; /* XXX: Compute this value? */
    
    __shared__ double sum[MAX_EQN];
    
    int2 *term = E->term + eqn*MAX_EQN;
    int2 pair = term[tid];
    int e = pair.x; /* 1 <= e <= E->lmesh.NNO */
    int i = pair.y; /* 0 <= i < MAX_EQN */
        
    if (i != -1) {
        /* XXX: Compute these values? */
        int eqn1 = E->ID[e].doff[1];
        int eqn2 = E->ID[e].doff[2];
        int eqn3 = E->ID[e].doff[3];
            
        higher_precision *B1, *B2, *B3;
        B1 = E->Eqn_k[1]+(e-1)*MAX_EQN;
        B2 = E->Eqn_k[2]+(e-1)*MAX_EQN;
        B3 = E->Eqn_k[3]+(e-1)*MAX_EQN;
        
        sum[tid] = B1[i]*E->temp[eqn1] +
                   B2[i]*E->temp[eqn2] +
                   B3[i]*E->temp[eqn3];
    } else {
        /* XXX: A considerable number of threads idle here. */
        sum[tid] = 0.0;
    }
    __syncthreads();
    
    /* Reduce the partial sums for this block.
       Based on reduce2() in the CUDA SDK. */
    for (unsigned int s = MAX_EQN/2; s > 0; s >>= 1) {
        if (tid < s) {
            sum[tid] += sum[tid + s];
        }
        /* XXX: not always necessary */
        __syncthreads();
    }
    
    if (tid == 0) {
        /* Each block writes one element of Ad... */
        Ad[eqn] += sum[0];
        /* ..and one element of d0. */
        d0[eqn] += E->temp[eqn];
    }
}

void do_gauss_seidel(
    struct Some_variables *E,
    double *d0,
    double *F, double *Ad,
    double acc,
    int *cycles,
    int guess
    )
{

    int count, steps;

    steps=*cycles;

    /* pointers to device memory */
    struct Some_variables *d_E = 0;
    double *d_d0 = 0, *d_F = 0, *d_Ad = 0;
    
    /* construct 'E' on the device */
    struct Some_variables s_E;
    construct_E(&d_E, &s_E, E);
    
    int neq = E->lmesh.NEQ;
    
    /* allocate memory on the device */
    hipMalloc((void**)&d_d0, (1+neq)*sizeof(double));
    hipMalloc((void**)&d_F, neq*sizeof(double));
    hipMalloc((void**)&d_Ad, (1+neq)*sizeof(double));
    
    /* copy input to the device */
    hipMemcpy(d_F, F, neq*sizeof(double), hipMemcpyHostToDevice);
    
    if (guess) {
        /* copy more input to the device */
        d0[E->lmesh.NEQ] = 0.0; /* normally done by n_assemble_del2_u() */
        hipMemcpy(d_d0, d0, (1+neq)*sizeof(double), hipMemcpyHostToDevice);
        
        dim3 block(MAX_EQN, 1, 1);
        dim3 grid(E->lmesh.NNO, NSD, 1);
        n_assemble_del2_u<<< grid, block >>>(d_E, d_d0, d_Ad, 1);
    
    } else {
        dim3 block(1, 1, 1);
        dim3 grid(E->lmesh.NEQ, 1, 1);
        gauss_seidel_0<<< grid, block >>>(d_E, d_d0, d_Ad);
    }
    
    for (count = 0; count < steps; ++count) {
        {
            dim3 block(1, 1, 1);
            dim3 grid(E->lmesh.NNO, NSD, 1);
            gauss_seidel_1<<< grid, block >>>(d_E, d_F, d_Ad);
            gauss_seidel_2<<< grid, block >>>(d_E, d_F, d_Ad);
        }
        
        /* Ad on boundaries differs after the following operation */
        {
            dim3 block(MAX_EQN, 1, 1);
            dim3 grid(E->lmesh.NNO, NSD, 1);
            gauss_seidel_3<<< grid, block >>>(d_E, d_d0, d_Ad);
        }
    }
    
    /* wait for completion */
    if (hipDeviceSynchronize() != hipSuccess) {
        assert(0 && "something went wrong");
    }
    
    /* copy output from device */
    hipMemcpy(Ad, d_Ad, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(d0, d_d0, (1+neq)*sizeof(double), hipMemcpyDeviceToHost);
    
    /* free device memory */
    hipFree(d_d0);
    hipFree(d_F);
    hipFree(d_Ad);
    
    destroy_E(d_E, &s_E);
    
    *cycles=count;
    
    return;
}


/*------------------------------------------------------------------------*/

static void assert_assumptions(struct All_variables *E, int level) {
    
    assert(E->control.NMULTIGRID);
    
    assert(E->sphere.caps_per_proc == CAPS_PER_PROC);
    
    assert(E->mesh.nsd == NSD);
    
    assert(E->parallel.nproc == 1);
}

static void collect_terms(
    struct Some_variables *E
    )
{
    /* Map out how to parallelize "Au[C[i]] += ..." and "Ad[C[j]] += ...". */
    
    const int neq = E->lmesh.NEQ;
    const int nno = E->lmesh.NNO;
    
    E->term = (int2 *)malloc((neq+1) * MAX_EQN * sizeof(int2));
    
    for (int e = 0; e <= neq; e++) {
        int2 *term = E->term + e*MAX_EQN;
        for (int j = 0; j < MAX_EQN; j++) {
            term[j].x = -1;
            term[j].y = -1;
        }
    }
    
    for (int e = 1; e <= nno; e++) {
        int *C = E->Node_map + (e-1)*MAX_EQN;
        for (int i = 0; i < MAX_EQN; i++) {
            int2 *term = E->term + C[i]*MAX_EQN;
            int j;
            for (j = 0; j < MAX_EQN; j++) {
                if (term[j].x == -1) {
                    term[j].x = e;
                    term[j].y = i;
                    break;
                }
            }
            assert(C[i] == neq || j < MAX_EQN);
        }
    }
    
    return;
}

extern "C" void gauss_seidel(
    struct All_variables *E,
    double **d0,
    double **F, double **Ad,
    double acc,
    int *cycles,
    int level,
    int guess
    )
{
    struct Some_variables kE;
    
    assert_assumptions(E, level);
    
    /* initialize 'Some_variables' with 'All_variables' */
    
    kE.lmesh.NEQ = E->lmesh.NEQ[level];
    kE.lmesh.NNO = E->lmesh.NNO[level];
    
    kE.ID    = E->ID[level][M];
    
    kE.Eqn_k[0] = 0;
    kE.Eqn_k[1] = E->Eqn_k1[level][M];
    kE.Eqn_k[2] = E->Eqn_k2[level][M];
    kE.Eqn_k[3] = E->Eqn_k3[level][M];
    kE.Node_map = E->Node_map[level][M];
    
    kE.BI = E->BI[level][M];
    
    kE.temp = E->temp[M];
    
    kE.NODE = E->NODE[level][M];
    
    collect_terms(&kE);
    
    do_gauss_seidel(
        &kE,
        d0[M],
        F[M], Ad[M],
        acc,
        cycles,
        guess
        );
}
