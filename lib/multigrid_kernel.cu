#include "hip/hip_runtime.h"
/* -*- C -*- */
/* vim:set ft=c: */

#if __CUDA_ARCH__ < 130
/* for double-precision floating-point */
#error This code requires compute capability 1.3 or higher; try giving "-arch sm_13".
#endif


#include "global_defs.h"
#include "element_definitions.h"
#include <assert.h>
#include <stdio.h>


enum {
    CAPS_PER_PROC = 1,
    M = 1, /* cap # */
    NSD = 3, /* Spatial extent: 3d */
    MAX_EQN = NSD*14,
};


struct Some_variables {
    int num_zero_resid;
    int *zero_resid;
    
    struct /*MESH_DATA*/ {
        int NEQ;
        int NNO;
    } lmesh;
    
    struct ID *ID;
    
    higher_precision *Eqn_k[NSD+1];
    int *Node_map;
    
    double *BI;
    
    double *temp;
    unsigned int *NODE;
    
    int2 **term;
};


/*------------------------------------------------------------------------*/
/* from Element_calculations.c */

__global__ void n_assemble_del2_u(
    struct Some_variables *E,
    double *u, double *Au,
    int strip_bcs
    )
{
    int n = blockIdx.x + 1; /* 1 <= n <= E->lmesh.NNO */
    int doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    unsigned int tid = threadIdx.x; /* 0 <= tid < MAX_EQN */
    
    /* Each block writes one element of Au in global memory: Au[eqn]. */
    int eqn = E->ID[n].doff[doff]; /* XXX: Compute this value? */
    
    if (strip_bcs) {
        /* See get_bcs_id_for_residual(). */
        unsigned int flags = E->NODE[n];
        unsigned int vb = 0x1 << doff; /* VBX, VBY, or VBZ */
        if (flags & vb) {
            /* no-op: Au[eqn] is zero */
            if (tid == 0) {
                Au[eqn] = 0.0;
            }
            /* XXX: Hundreds of blocks exit here (E->num_zero_resid).
               Does it matter? */
            return;
        }
    }
    
    /* The partial sum computed by this thread. */
    double acc;
    
    /* Part I: The terms here are easily derived from the block and
       thread indices. */
    {
        int e = n; /* 1 <= e <= E->lmesh.NNO */
        int i = (int)tid; /* 0 <= i < MAX_EQN */
        
        if (i < 3) {
            acc = 0.0;
        } else {
            int *C = E->Node_map + (e-1)*MAX_EQN;
            higher_precision *B = E->Eqn_k[doff]+(e-1)*MAX_EQN;
            double UU = u[C[i]];
            acc = B[i]*UU;
        }
    }
    
    /* Part II: These terms are more complicated. */
    {
        int2 pair = E->term[eqn][tid];
        int e = pair.x; /* 1 <= e <= E->lmesh.NNO */
        int i = pair.y; /* 0 <= i < MAX_EQN */
        
        if (i != -1) {
            /* XXX: Compute these values? */
            int eqn1 = E->ID[e].doff[1];
            int eqn2 = E->ID[e].doff[2];
            int eqn3 = E->ID[e].doff[3];
            
            double U1 = u[eqn1];
            double U2 = u[eqn2];
            double U3 = u[eqn3];
            
            higher_precision *B1, *B2, *B3;
            B1 = E->Eqn_k[1]+(e-1)*MAX_EQN;
            B2 = E->Eqn_k[2]+(e-1)*MAX_EQN;
            B3 = E->Eqn_k[3]+(e-1)*MAX_EQN;
            
            acc += B1[i]*U1 +
                   B2[i]*U2 +
                   B3[i]*U3;
        } else {
            /* XXX: A considerable number of threads idle here. */
        }
    }
    
    /* Reduce the partial sums for this block.
       Based on reduce2() in the CUDA SDK. */
    __shared__ double sum[MAX_EQN];
    sum[tid] = acc;
    __syncthreads();
    for (unsigned int s = MAX_EQN/2; s > 0; s >>= 1) {
        if (tid < s) {
            sum[tid] += sum[tid + s];
        }
        /* XXX: not always necessary */
        __syncthreads();
    }
    
    /* Each block writes one element of Au in global memory. */
    if (tid == 0) {
        Au[eqn] = sum[0];
    }
    
    return;
}


/*------------------------------------------------------------------------*/
/* These are based on the function from General_matrix_functions.c. */

__global__ void gauss_seidel_0(
    struct Some_variables *E,
    double *d0,
    double *Ad
    )
{
    const double zeroo = 0.0;
    int i;
    
    i = blockIdx.x; /* 0 <= i < E->lmesh.NEQ */
    d0[i] = Ad[i] = zeroo;
}

__global__ void gauss_seidel_1(
    struct Some_variables *E,
    double *F, double *Ad
    )
{
    const double zeroo = 0.0;
    const int neq = E->lmesh.NEQ;
    
    int i, doff, eqn;
    
    i = blockIdx.x + 1; /* 1 <= i <= E->lmesh.NNO */
    doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    eqn = E->ID[i].doff[doff];
    
    if (E->NODE[i] & OFFSIDE) {
        E->temp[eqn] = (F[eqn] - Ad[eqn])*E->BI[eqn];
    } else {
        E->temp[eqn] = zeroo;
    }
    
    if (i == 1 && doff == 1) {
        E->temp[neq] = zeroo;
        Ad[neq] = zeroo;
    }
}

__global__ void gauss_seidel_2(
    struct Some_variables *E,
    double *F, double *Ad
    )
{
    int i, doff, eqn;
    
    i = blockIdx.x + 1; /* 1 <= i <= E->lmesh.NNO */
    doff = blockIdx.y + 1; /* 1 <= doff < NSD */ 
    eqn = E->ID[i].doff[doff];
    
    int *C;
    higher_precision *B;
    double UU, Ad_eqn;
    int j;
    
    C = E->Node_map+(i-1)*MAX_EQN;
    B = E->Eqn_k[doff]+(i-1)*MAX_EQN;
    
    /* load from global memory */
    Ad_eqn = Ad[eqn];
    
    /* Ad on boundaries differs after the following operation, but
       no communications are needed yet, because boundary Ad will
       not be used for the G-S iterations for interior nodes */
    
    for (j=3;j<MAX_EQN;j++)  {
        UU = E->temp[C[j]];
        Ad_eqn += B[j]*UU;
    }
    
    /* store to global memory */
    Ad[eqn] = Ad_eqn;
    
    if (!(E->NODE[i] & OFFSIDE))   {
        E->temp[eqn] = (F[eqn] - Ad_eqn)*E->BI[eqn];
    }

}

void do_gauss_seidel(
    struct Some_variables *E,
    double *d0,
    double *F, double *Ad,
    double acc,
    int *cycles,
    int guess
    )
{

    int count,i,j,steps;
    int *C;
    int eqn1,eqn2,eqn3;

    higher_precision *B1,*B2,*B3;

    steps=*cycles;

    dim3 neqBlock(1, 1, 1);
    dim3 neqGrid(E->lmesh.NEQ, 1, 1);
    
    dim3 nnoBlock(1, 1, 1);
    dim3 nnoGrid(E->lmesh.NNO, NSD, 1);
    
    /* XXX: allocate & init device memory */
    struct Some_variables *d_E = 0;
    double *d_d0 = 0, *d_F = 0, *d_Ad = 0;
    
    if (guess) {
        /* XXX */
        d_Ad[E->lmesh.NEQ] = 0.0; /* Au -- unnecessary? */
        d_d0[E->lmesh.NEQ] = 0.0; /* u */
        
        dim3 block(MAX_EQN, 1, 1);
        dim3 grid(E->lmesh.NNO, NSD, 1);
        n_assemble_del2_u<<< grid, block >>>(d_E, d_d0, d_Ad, 1);
    } else {
        gauss_seidel_0<<< neqGrid, neqBlock >>>(d_E, d_d0, d_Ad);
    }
    
    for (count = 0; count < steps; ++count) {
        
        gauss_seidel_1<<< nnoGrid, nnoBlock >>>(d_E, d_F, d_Ad);
        gauss_seidel_2<<< nnoGrid, nnoBlock >>>(d_E, d_F, d_Ad);
        
        
        /* XXX: How to parallelize this? */
        for (i=1;i<=E->lmesh.NNO;i++) {

            /* Ad on boundaries differs after the following operation */
            for (j=0;j<MAX_EQN;j++) {
                Ad[C[j]]  += B1[j]*E->temp[eqn1]
                             +  B2[j]*E->temp[eqn2]
                             +  B3[j]*E->temp[eqn3];
            }

            d0[eqn1] += E->temp[eqn1];
            d0[eqn2] += E->temp[eqn2];
            d0[eqn3] += E->temp[eqn3];
        }
    }
    
    /* wait for completion */
    hipDeviceSynchronize();
    
    *cycles=count;
    return;
}


/*------------------------------------------------------------------------*/

static void assert_assumptions(struct All_variables *E, int level) {
    
    assert(E->control.NMULTIGRID);
    
    assert(E->sphere.caps_per_proc == CAPS_PER_PROC);
    
    assert(E->mesh.nsd == NSD);
    
    assert(E->parallel.nproc == 1);
}

static void tally_n_assemble_del2_u(
    struct Some_variables *E //,
    //double *u, double *Au,
    //int strip_bcs
    )
{
    int e,i;
    int eqn1,eqn2,eqn3;
    
#if 0
    double UU,U1,U2,U3;
#endif

    int *C;
#if 0
    higher_precision *B1,*B2,*B3;
#endif

    const int neq=E->lmesh.NEQ;
    const int nno=E->lmesh.NNO;
    
    /*
     * Au = E->Eqn_k? * u
     *  where E->Eqn_k? is the sparse stiffness matrix
     */
    
    int maxAcc, total;
    int *tally;
    int **threadMap, **threadTally;
    int2 **terms;
    int f;
    
    tally = (int *)malloc((neq+1) * sizeof(int));
    threadMap = (int **)malloc((neq+1)* sizeof(int*));
    threadTally = (int **)malloc((neq+1)* sizeof(int*));
    terms = (int2 **)malloc((neq+1)* sizeof(int2 *));
    
    for(e=0;e<=neq;e++) {
        //Au[e]=0.0;
        tally[e] = 0;
        threadMap[e] = (int *)malloc((neq+1) * sizeof(int));
        threadTally[e] = (int *)malloc((neq+1) * sizeof(int));
        terms[e] = (int2 *)malloc((MAX_EQN+1) * sizeof(int2));
        for(f=0;f<=neq;f++) {
            threadMap[e][f] = -1;
            threadTally[e][f] = 0;
        }
        for (f = 0; f < MAX_EQN; ++f) {
            terms[e][f].x = -1;
            terms[e][f].y = -1;
        }
        terms[e][MAX_EQN].x = 0;
        terms[e][MAX_EQN].y = 0;
    }

#if 0
    u[neq] = 0.0;
#endif

    for(e=1;e<=nno;e++)     {

        eqn1=E->ID[e].doff[1];
        eqn2=E->ID[e].doff[2];
        eqn3=E->ID[e].doff[3];
        
        /* could compute, but 'Node_map' is more complicated */
        assert(eqn1 == 3*(e-1));
        assert(eqn2 == eqn1+1);
        assert(eqn3 == eqn1+2);
        
        /* could put maps in constant memory */
        
        /*
         * Key observation: after parallelizing on 'e' (either one):
         *
         *   ID[e].doff[1,2,3]
         *   C
         *
         * are fixed for each thread.  => Not worth obsessing over?
         */
        
        /*
         * Put Au[eqnX] into shared memory; it is accessed almost MAX_EQN=42 times.
         *
         * "Au[e]=0.0" should be unnecessary -- single write at end of fn
         * from 'AuX' local var.. so actually, Au[eqnX] sits in register
         * 
         * But what about "Au[C[i]]"???????????
         */
        
        /*
         * neq vs. nno
         *
         * neq == 3*nno
         * warp=32; only 2 threads wasted
         * better: 3 warps 32*3
         *
         * use y for "dimension index"? (block size 32x3; nno % 32)
         */
        
#if 0
        U1 = u[eqn1];
        U2 = u[eqn2];
        U3 = u[eqn3];
#endif

        C=E->Node_map + (e-1)*MAX_EQN;
#if 0
        B1=E->Eqn_k[1]+(e-1)*MAX_EQN;
        B2=E->Eqn_k[2]+(e-1)*MAX_EQN;
        B3=E->Eqn_k[3]+(e-1)*MAX_EQN;
#endif

        for(i=3;i<MAX_EQN;i++)  {
#if 0
            UU = u[C[i]];
            Au[eqn1] += B1[i]*UU;
            Au[eqn2] += B2[i]*UU;
            Au[eqn3] += B3[i]*UU;
#endif
            ++tally[eqn1];
            ++tally[eqn2];
            ++tally[eqn3];
            for(f=0;f<=neq;f++) {
                if (threadMap[eqn1][f] == e) {
                    ++threadTally[eqn1][f];
                    break;
                }
                if (threadMap[eqn1][f] == -1) {
                    threadMap[eqn1][f] = e;
                    ++threadTally[eqn1][f];
                    break;
                }
            }
            for(f=0;f<=neq;f++) {
                if (threadMap[eqn2][f] == e) {
                    ++threadTally[eqn2][f];
                    break;
                }
                if (threadMap[eqn2][f] == -1) {
                    threadMap[eqn2][f] = e;
                    ++threadTally[eqn2][f];
                    break;
                }
            }
            for(f=0;f<=neq;f++) {
                if (threadMap[eqn3][f] == e) {
                    ++threadTally[eqn3][f];
                    break;
                }
                if (threadMap[eqn3][f] == -1) {
                    threadMap[eqn3][f] = e;
                    ++threadTally[eqn3][f];
                    break;
                }
            }
        }
        for(i=0;i<MAX_EQN;i++) {
#if 0
            Au[C[i]] += B1[i]*U1+B2[i]*U2+B3[i]*U3;
#endif
            ++tally[C[i]];
            for(f=0;f<=neq;f++) {
                if (threadMap[C[i]][f] == e) {
                    ++threadTally[C[i]][f];
                    break;
                }
                if (threadMap[C[i]][f] == -1) {
                    threadMap[C[i]][f] = e;
                    ++threadTally[C[i]][f];
                    break;
                }
            }
            ++terms[C[i]][MAX_EQN].y;
            for (f = 0; f < MAX_EQN; ++f) {
                if (terms[C[i]][f].y == -1) {
                    terms[C[i]][f].x = e;
                    terms[C[i]][f].y = i;
                    break;
                }
            }
            assert(C[i] == neq || f < MAX_EQN);
        }

    }     /* end for e */
    
    maxAcc = 0;
    total = 0;
    for(e=0;e<=neq;e++) {
        int myTally;
        fprintf(stderr, "Au[%d]: %d times", e, tally[e]);
        if (e < neq)
            maxAcc = max(maxAcc, tally[e]);
        total += tally[e];
        myTally = 0;
        for(f=0;f<=neq;f++) {
            if (threadMap[e][f] == -1)
                break;
            fprintf(stderr, " %d(%d)", threadMap[e][f], threadTally[e][f]);
            myTally += threadTally[e][f];
        }
        fprintf(stderr, " (%d times)\n", myTally);
    }
    //fprintf(stderr, "Au[%d] == %f\n", e - 1, Au[e]);
    fprintf(stderr, "max accesses %d\n", maxAcc);
    fprintf(stderr, "total accesses %d\n", total);
    
    fprintf(stderr, "\nterms:\n");
    for(e=0;e<=neq;e++) {
        fprintf(stderr, "Au[%d]: %d terms %s", e, terms[e][MAX_EQN].y,
                terms[e][MAX_EQN].y > MAX_EQN ? "XXXTO" : "");
        for (f = 0; f < MAX_EQN; ++f) {
            if (terms[e][f].y == -1)
                break;
            fprintf(stderr, " %d(%d)", terms[e][f].y, terms[e][f].x);
        }
        fprintf(stderr, "\n");
    }
    
#if 0
    if (strip_bcs)
        strip_bcs_from_residual(E,Au);
#endif

    return;
}

extern "C" void gauss_seidel(
    struct All_variables *E,
    double **d0,
    double **F, double **Ad,
    double acc,
    int *cycles,
    int level,
    int guess
    )
{
    struct Some_variables kE;
    
    assert_assumptions(E, level);
    
    /* initialize 'Some_variables' with 'All_variables' */
    
    kE.num_zero_resid = E->num_zero_resid[level][M];
    kE.zero_resid = E->zero_resid[level][M];
    
    kE.lmesh.NEQ = E->lmesh.NEQ[level];
    kE.lmesh.NNO = E->lmesh.NNO[level];
    
    kE.ID    = E->ID[level][M];
    
    kE.Eqn_k[0] = 0;
    kE.Eqn_k[1] = E->Eqn_k1[level][M];
    kE.Eqn_k[2] = E->Eqn_k2[level][M];
    kE.Eqn_k[3] = E->Eqn_k3[level][M];
    kE.Node_map = E->Node_map[level][M];
    
    kE.BI = E->BI[level][M];
    
    kE.temp = E->temp[M];
    
    kE.NODE = E->NODE[level][M];
    
                                       /* XXX */
    do {
        int i, doff, print;
        for (i=1;i<=kE.lmesh.NNO;i++) {
            print = (i < 10 || i > kE.lmesh.NNO - 10);
            if (print)
                fprintf(stderr, "%04d:", i);
            for (doff = 1; doff <= 3; ++doff) {
                assert(kE.ID[i].doff[doff] == /*NSD*/ 3 * (i - 1) + doff - 1);
                if (print)
                    fprintf(stderr, " %d", kE.ID[i].doff[doff]);
            }
            if (print)
                fprintf(stderr, "\n");
        }
        fprintf(stderr, "\n0 - NEQ %d\n", kE.lmesh.NEQ);
    } while (0);
    tally_n_assemble_del2_u(&kE);
    do {
        int i;
        fprintf(stderr, "E->num_zero_resid == %d\n", kE.num_zero_resid);
        for (i=1;i<=kE.num_zero_resid;i++)
            fprintf(stderr, "    Au[%d] = 0.0\n", kE.zero_resid[i]);
    } while (0);
    assert(0);
}
